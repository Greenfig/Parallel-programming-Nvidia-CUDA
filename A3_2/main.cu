#include <iostream>
#include <thread>
#include <iomanip>
#include <cstdlib>
#include <hip/hip_runtime.h>
// to remove intellisense highlighting
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include "BruteFactor.h"

using namespace std;
void brute(const char* a, BruteFactor** LN){
	*LN = new BruteFactor(a);
	(*LN)->getPrimes(); //Brute force
}
void brute2(const char* a, BruteFactor** LN){
	*LN = new BruteFactor(a);
	(*LN)->getPrimesCuda(); //Brute force
}


int main(){
	//Change this to your project path
	//string filename, a_path = "C:\\Users\\Rene\\ReneA-GDrive\\Project\\DPS915\\A3_2\\A3_2\\";
	string filename, a_path = "D:\\ReneA-GDrive\\Project\\DPS915\\A3_2\\A3_2\\";

	do{
		cout << "Enter File Name : ";
		cin >> filename;
		if (filename.compare("exit") != 0){
			//use for small numbers < 20 digits
			BruteFactor *noCuda, *hasCuda;
			string location = (a_path + filename);
			thread /*t1(brute, location.c_str(), &noCuda),*/ t2(brute2, location.c_str(), &hasCuda);
			//t1.join();
			t2.join();
			//noCuda->display();
			hasCuda->display();
		}
	} while (filename.compare("exit"));
	return 0;

}
